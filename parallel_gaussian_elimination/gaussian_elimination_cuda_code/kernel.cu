#include "hip/hip_runtime.h"
﻿// Eliminacja Gaussa z wykorzystaniem CUDA 


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>


void gaussianEliminationWithCuda(double* matrix, int size);
double* readMatrix(char* file_addr);
int readLength(char* file_addr);
void findSolution(double* matrix, int size);



__global__ void operationA(double* matrix, double* multipliers, int round_number, int height)
{
    int i = threadIdx.x;
    int j = height - 1 - i;
    int len = height + 1;

    multipliers[j] = matrix[j * len + round_number] / matrix[round_number * len + round_number];
}


__global__ void operationB(double* matrix, double* multipliers, double* substractors, int round_number, int height)
{
    int i = threadIdx.x;
    int j = height - 1 - i;
    int len = height + 1;

    for (int pos = 0; pos < len; pos++) {
        substractors[j * len + pos] = matrix[round_number * len + pos] * multipliers[j];
    }
}


__global__ void operationC(double* matrix, double* substractors, int round_number, int height)
{
    int i = threadIdx.x;
    int j = height - 1 - i;
    int len = height + 1;

    for (int pos = 0; pos < len; pos++) {
        matrix[j * len + pos] = matrix[j * len + pos] - substractors[j * len + pos];
    }
}



int main()
{
    printf("___Gaussian elimination with CUDA___\n");

    char* file_link = "file1.txt";
    int size = readLength(file_link);
    double* matrix = readMatrix(file_link);

    printf("\nGiven matrix:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j <= size; j++) {
            printf("%f ", matrix[i * (size + 1) + j]);
        }
        printf("\n");
    }

    gaussianEliminationWithCuda(matrix, size);

    printf("\nMatrix after Gaussian elimination:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j <= size; j++) {
            printf("%f ", matrix[i * (size + 1) + j]);
        }
        printf("\n");
    }

    findSolution(matrix, size);

    printf("\nFound solution:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j <= size; j++) {
            printf("%f ", matrix[i * (size + 1) + j]);
        }
        printf("\n");
    }


    char* result_file_link = "file2.txt";
    double* matrix_exp = readMatrix(result_file_link);


    printf("\nExpected solution:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j <= size; j++) {
            printf("%f ", matrix_exp[i * (size + 1) + j]);
        }
        printf("\n");
    }

    return 0;
}

void gaussianEliminationWithCuda(double* matrix, int size)
{
    hipSetDevice(0);

    int len = size + 1;
    int i, j, to_change;
    double copied;
    double* substractors = (double*)calloc(size * (size + 1), sizeof(double));
    double* multipliers = (double*)calloc(size, sizeof(double));

    double* gpu_matrix = 0;
    double* gpu_A_mulitpliers = 0;
    double* gpu_B_substractor = 0;

    hipMalloc((void**)&gpu_matrix, size * len * sizeof(double));
    hipMalloc((void**)&gpu_A_mulitpliers, size * sizeof(double));
    hipMalloc((void**)&gpu_B_substractor, size * len * sizeof(double));


    hipMemcpy(gpu_matrix, matrix, size * (size + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_A_mulitpliers, multipliers, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_B_substractor, substractors, size * (size + 1) * sizeof(double), hipMemcpyHostToDevice);

    for (int round = 1; round < size; round++) {
        operationA << <1, (size - round) >> > (gpu_matrix, gpu_A_mulitpliers, round - 1, size);
        hipDeviceSynchronize();
        operationB << <1, (size - round) >> > (gpu_matrix, gpu_A_mulitpliers, gpu_B_substractor, round - 1, size);
        hipDeviceSynchronize();
        operationC << <1, (size - round) >> > (gpu_matrix, gpu_B_substractor, round - 1, size);
        hipDeviceSynchronize();


        hipMemcpy(matrix, gpu_matrix, size * (size + 1) * sizeof(double), hipMemcpyDeviceToHost);

        i = round - 1;
        if (matrix[len * i + i] == 0.0) {
            to_change = i;
            for (j = i + 1; j < size; j++) {
                if (matrix[len * j + i] != 0.0) {
                    to_change = j;
                    break;
                }

            }

            for (j = 0; j < len; j++) {
                matrix[len * i + j], matrix[len * to_change + j] = matrix[len * to_change + j], matrix[len * i + j];
            }

        }

        hipMemcpy(gpu_matrix, matrix, size * (size + 1) * sizeof(double), hipMemcpyHostToDevice);




    }

    hipMemcpy(matrix, gpu_matrix, size * (size + 1) * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(gpu_matrix);
    hipFree(gpu_A_mulitpliers);
    hipFree(gpu_B_substractor);

    free(substractors);
    free(multipliers);

    hipDeviceReset();

    return;
}



double* readMatrix(char* file_addr)
{
    double* matrix = 0;
    int pos_in_matrix = 0;

    int max_len = 20;
    FILE* ptr;
    char ch;

    double curr_val;
    int size = -1;
    int position = 0;
    int row_counter = 0;
    int last_col_counter = 0;

    char* buffer = (char*)calloc(max_len, sizeof(char));

    ptr = fopen(file_addr, "r");

    if (NULL == ptr) {
        printf("FILE ERROR \n");
    }

    do {
        ch = fgetc(ptr);
        if (ch != ' ' && ch != '\n' && ch != EOF) {
            buffer[position] = ch;
            position++;
        }
        else {
            if (size == -1) {
                size = atoi(buffer);
                matrix = (double*)calloc(size * (size + 1), sizeof(double));
            }
            else {
                curr_val = atof(buffer);
                if (row_counter < size) {
                    matrix[pos_in_matrix] = curr_val;
                    pos_in_matrix++;
                    if (pos_in_matrix % (size + 1) == size) {
                        pos_in_matrix++;
                    }
                }
                else {
                    matrix[last_col_counter * (size + 1) + size] = curr_val;
                    last_col_counter++;
                }

                if (ch == '\n') {
                    row_counter++;
                }
            }
            position = 0;
            memset(buffer, '0', max_len);
        }
    } while (ch != EOF);

    fclose(ptr);

    return matrix;
}


int readLength(char* file_addr)
{
    int max_len = 10;
    FILE* ptr;
    char ch;

    int size = -1;

    char* buffer = (char*)calloc(max_len, sizeof(char));
    int position = 0;

    ptr = fopen(file_addr, "r");

    if (NULL == ptr) {
        printf("FILE ERROR \n");
    }

    do {
        ch = fgetc(ptr);
        if (ch != ' ' && ch != '\n' && ch != EOF) {
            buffer[position] = ch;
            position++;
        }
        else {
            if (size == -1) {
                size = atoi(buffer);
                return size;
            }
        }
    } while (ch != EOF);
    return 0;
}


void findSolution(double* matrix, int size) {
    int len = size + 1;
    double curr_sum;

    for (int i = size - 1; i >= 0; i--) {
        curr_sum = matrix[len * i + size];
        for (int j = i + 1; j < size; j++) {
            curr_sum -= matrix[len * i + j] * matrix[len * j + size];
            matrix[len * i + j] = 0.0;
        }
        matrix[len * i + size] = curr_sum / matrix[len * i + i];
        matrix[len * i + i] = 1.0;
    }

    return;
}